#include "hip/hip_runtime.h"
#include "NDArray.h"
#include <iostream>

__global__ void copy_test(int* input, int xsize, int ysize, int* output) {
    int tile_x = blockIdx.x;
    int tile_y = blockIdx.y;

    int x = tile_x * blockDim.x + threadIdx.x;
    int y = tile_y * blockDim.y + threadIdx.y;

    int offset = x * ysize + y;
    if (x < xsize && y < ysize) {
        output[offset] = input[offset];
    }
}

int main() {

    NDArray<int> input({76, 899});
    NDArray<int> output({76, 899});

    input.device_allocate();
    output.device_allocate();

    input.initialize(5);
    input(0, 1) = 10;
    input(1, 0) = 1000;
    input(75, 888) = 7;
    input(0, 0) = 9;

    input.copy_to_device();

    LaunchConfig l = get_tile_launch_config({8, 8},
                                {input.strides[0], input.strides[1]});

    double time = benchmark(1, 1, [&]() {
    copy_test<<<l.block_config, l.thread_config>>>(input.dev_alloc.get(),
                                                   input.strides[0],
                                                   input.strides[1],
                                                   output.dev_alloc.get());
    });

    std::cout << "Time: " << time << "ms" << std::endl;
    output.copy_from_device();

    for (int x = 0; x < input.strides[0];  x++) {
        for (int y = 0; y < input.strides[1]; y++) {
            if (output(x, y) != input(x, y)) {
                std::cerr << "Test failed" << std::endl;
                exit(-1);
            }
        }
    }
    std::cout << "Test passed" << std::endl;
}
